#include "hip/hip_runtime.h"
/**
 * jacobi2D.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Will Killian <killian@udel.edu>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */

#include <stdio.h>
#include <unistd.h>
#include <time.h>
#include <sys/time.h>
#include <string.h>
#include <stdlib.h>
#include <stdarg.h>
#include <math.h>

#define POLYBENCH_TIME 1

#include "jacobi2D.cuh"
#include "../../utilities/polybench.h"
#include "../../utilities/polybenchUtilFuncts.h"
#include "../../utilities/gputimer.h"

// define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 0.05

// define perforation rates
#define LOOP_PERFORATION_RATE 1.0
#define KERNEL_LAUNCH_LOOP_RATE 0.85
#define GRID_PERFORATION_RATE 1.0
#define BLOCK_PERFORATION_RATE 1.0

#define RUN_ON_CPU

void init_array(int n, DATA_TYPE POLYBENCH_2D(A, N, N, n, n), DATA_TYPE POLYBENCH_2D(B, N, N, n, n))
{
    int i, j;

    for (i = 0; i < n; i++)
    {
        for (j = 0; j < n; j++)
        {
            A[i][j] = ((DATA_TYPE)i * (j + 2) + 10) / N;
            B[i][j] = ((DATA_TYPE)(i - 4) * (j - 1) + 11) / N;
        }
    }
}

void runJacobi2DCpu(int tsteps, int n, DATA_TYPE POLYBENCH_2D(A, N, N, n, n), DATA_TYPE POLYBENCH_2D(B, N, N, n, n))
{
    for (int t = 0; t < _PB_TSTEPS; t++)
    {
        for (int i = 1; i < _PB_N - 1; i++)
        {
            for (int j = 1; j < _PB_N - 1; j++)
            {
                B[i][j] = 0.2f * (A[i][j] + A[i][(j - 1)] + A[i][(1 + j)] + A[(1 + i)][j] + A[(i - 1)][j]);
            }
        }

        for (int i = 1; i < _PB_N - 1; i++)
        {
            for (int j = 1; j < _PB_N - 1; j++)
            {
                A[i][j] = B[i][j];
            }
        }
    }
}

__global__ void runJacobiCUDA_kernel1(int n, DATA_TYPE *A, DATA_TYPE *B)
{
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if ((i >= 1) && (i < (_PB_N - 1)) && (j >= 1) && (j < (_PB_N - 1)))
    {
        B[i * N + j] = 0.2f * (A[i * N + j] + A[i * N + (j - 1)] + A[i * N + (1 + j)] + A[(1 + i) * N + j] + A[(i - 1) * N + j]);
    }
}

__global__ void runJacobiCUDA_kernel2(int n, DATA_TYPE *A, DATA_TYPE *B)
{
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if ((i >= 1) && (i < (_PB_N - 1)) && (j >= 1) && (j < (_PB_N - 1)))
    {
        A[i * N + j] = B[i * N + j];
    }
}

void compareResults(int n, DATA_TYPE POLYBENCH_2D(a, N, N, n, n), DATA_TYPE POLYBENCH_2D(a_outputFromGpu, N, N, n, n), DATA_TYPE POLYBENCH_2D(b, N, N, n, n), DATA_TYPE POLYBENCH_2D(b_outputFromGpu, N, N, n, n))
{
    int i, j, fail, total;
    fail = 0;
    total = 0;

    // Compare output from CPU and GPU
    for (i = 0; i < n; i++)
    {
        for (j = 0; j < n; j++)
        {
            total++;
            if (percentDiff(a[i][j], a_outputFromGpu[i][j]) > PERCENT_DIFF_ERROR_THRESHOLD)
            {
                fail++;
            }
        }
    }

    for (i = 0; i < n; i++)
    {
        for (j = 0; j < n; j++)
        {
            total++;
            if (percentDiff(b[i][j], b_outputFromGpu[i][j]) > PERCENT_DIFF_ERROR_THRESHOLD)
            {
                fail++;
            }
        }
    }

    // Print results
    printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f Percent: %d\n", PERCENT_DIFF_ERROR_THRESHOLD, fail);

    printf("Total number of comparations: %d\n", total);
    printf("Loop perforation rate: %f\n", LOOP_PERFORATION_RATE);
    printf("Block perforation rate: %f\n", BLOCK_PERFORATION_RATE);
}

void runJacobi2DCUDA(int tsteps, int n, DATA_TYPE POLYBENCH_2D(A, N, N, n, n), DATA_TYPE POLYBENCH_2D(B, N, N, n, n), DATA_TYPE POLYBENCH_2D(A_outputFromGpu, N, N, n, n), DATA_TYPE POLYBENCH_2D(B_outputFromGpu, N, N, n, n))
{
    DATA_TYPE *Agpu;
    DATA_TYPE *Bgpu;

    hipMalloc(&Agpu, N * N * sizeof(DATA_TYPE));
    hipMalloc(&Bgpu, N * N * sizeof(DATA_TYPE));
    hipMemcpy(Agpu, A, N * N * sizeof(DATA_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(Bgpu, B, N * N * sizeof(DATA_TYPE), hipMemcpyHostToDevice);

    dim3 block(ceil(DIM_THREAD_BLOCK_X * BLOCK_PERFORATION_RATE), ceil(DIM_THREAD_BLOCK_Y * BLOCK_PERFORATION_RATE));
    dim3 grid((unsigned int)ceil(((float)N) / ((float)block.x) * GRID_PERFORATION_RATE), (unsigned int)ceil(((float)N) / ((float)block.y) * GRID_PERFORATION_RATE));

    /* Start timer. */
    GpuTimer gpuTimer;
    gpuTimer.Start();

    for (int t = 0; t < _PB_TSTEPS * KERNEL_LAUNCH_LOOP_RATE; t++)
    {
        runJacobiCUDA_kernel1<<<grid, block>>>(n, Agpu, Bgpu);
        hipDeviceSynchronize();
        runJacobiCUDA_kernel2<<<grid, block>>>(n, Agpu, Bgpu);
        hipDeviceSynchronize();
    }

    /* Stop and print timer. */
    gpuTimer.Stop();
    float elapsed_time = gpuTimer.Elapsed() / 1000;
    printf("GPU Time in seconds:\n");
    printf("%f\n", elapsed_time);

    hipMemcpy(A_outputFromGpu, Agpu, sizeof(DATA_TYPE) * N * N, hipMemcpyDeviceToHost);
    hipMemcpy(B_outputFromGpu, Bgpu, sizeof(DATA_TYPE) * N * N, hipMemcpyDeviceToHost);

    hipFree(Agpu);
    hipFree(Bgpu);
}

/* DCE code. Must scan the entire live-out data.
   Can be used also to check the correctness of the output. */
static void print_array(int n,
                        DATA_TYPE POLYBENCH_2D(A, N, N, n, n))

{
    int i, j;

    for (i = 0; i < n; i++)
        for (j = 0; j < n; j++)
        {
            fprintf(stderr, DATA_PRINTF_MODIFIER, A[i][j]);
            if ((i * n + j) % 20 == 0)
                fprintf(stderr, "\n");
        }
    fprintf(stderr, "\n");
}

int main(int argc, char **argv)
{
    /* Retrieve problem size. */
    int n = N;
    int tsteps = TSTEPS;

    POLYBENCH_2D_ARRAY_DECL(a, DATA_TYPE, N, N, n, n);
    POLYBENCH_2D_ARRAY_DECL(b, DATA_TYPE, N, N, n, n);
    POLYBENCH_2D_ARRAY_DECL(a_outputFromGpu, DATA_TYPE, N, N, n, n);
    POLYBENCH_2D_ARRAY_DECL(b_outputFromGpu, DATA_TYPE, N, N, n, n);

    init_array(n, POLYBENCH_ARRAY(a), POLYBENCH_ARRAY(b));
    runJacobi2DCUDA(tsteps, n, POLYBENCH_ARRAY(a), POLYBENCH_ARRAY(b), POLYBENCH_ARRAY(a_outputFromGpu), POLYBENCH_ARRAY(b_outputFromGpu));

#ifdef RUN_ON_CPU

    /* Start timer. */
    polybench_start_instruments;

    runJacobi2DCpu(tsteps, n, POLYBENCH_ARRAY(a), POLYBENCH_ARRAY(b));

    /* Stop and print timer. */
    printf("CPU Time in seconds:\n");
    polybench_stop_instruments;
    polybench_print_instruments;

    compareResults(n, POLYBENCH_ARRAY(a), POLYBENCH_ARRAY(a_outputFromGpu), POLYBENCH_ARRAY(b), POLYBENCH_ARRAY(b_outputFromGpu));

#else // prevent dead code elimination

    polybench_prevent_dce(print_array(n, POLYBENCH_ARRAY(a_outputFromGpu)));

#endif // RUN_ON_CPU

    POLYBENCH_FREE_ARRAY(a);
    POLYBENCH_FREE_ARRAY(a_outputFromGpu);
    POLYBENCH_FREE_ARRAY(b);
    POLYBENCH_FREE_ARRAY(b_outputFromGpu);

    return 0;
}

#include "../../utilities/polybench.c"