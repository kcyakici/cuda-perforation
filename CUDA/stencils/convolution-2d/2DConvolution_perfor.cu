#include "hip/hip_runtime.h"
/**
 * 2DConvolution.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Will Killian <killian@udel.edu>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */

#include <unistd.h>
#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include <stdlib.h>
#include <stdarg.h>
#include <string.h>
#include <hip/hip_runtime.h>

#define POLYBENCH_TIME 1

#include "2DConvolution.cuh"
#include "../../utilities/polybench.h"
#include "../../utilities/polybenchUtilFuncts.h"
#include "../../utilities/gputimer.h"

// define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 0.05

// define perforation rates
#define LOOP_PERFORATION_RATE 1.0
#define GRID_PERFORATION_RATE 1.0
#define BLOCK_PERFORATION_RATE 1.0

#define GPU_DEVICE 0

#define RUN_ON_CPU

void conv2D(int ni, int nj, DATA_TYPE POLYBENCH_2D(A, NI, NJ, ni, nj), DATA_TYPE POLYBENCH_2D(B, NI, NJ, ni, nj))
{
    int i, j;
    DATA_TYPE c11, c12, c13, c21, c22, c23, c31, c32, c33;

    c11 = +0.2;
    c21 = +0.5;
    c31 = -0.8;
    c12 = -0.3;
    c22 = +0.6;
    c32 = -0.9;
    c13 = +0.4;
    c23 = +0.7;
    c33 = +0.10;

    for (i = 1; i < _PB_NI - 1; ++i) // 0
    {
        for (j = 1; j < _PB_NJ - 1; ++j) // 1
        {
            B[i][j] = c11 * A[(i - 1)][(j - 1)] + c12 * A[(i + 0)][(j - 1)] + c13 * A[(i + 1)][(j - 1)] + c21 * A[(i - 1)][(j + 0)] + c22 * A[(i + 0)][(j + 0)] + c23 * A[(i + 1)][(j + 0)] + c31 * A[(i - 1)][(j + 1)] + c32 * A[(i + 0)][(j + 1)] + c33 * A[(i + 1)][(j + 1)];
        }
    }
}

void init(int ni, int nj, DATA_TYPE POLYBENCH_2D(A, NI, NJ, ni, nj))
{
    int i, j;

    for (i = 0; i < ni; ++i)
    {
        for (j = 0; j < nj; ++j)
        {
            A[i][j] = (float)rand() / RAND_MAX;
        }
    }
}

void compareResults(int ni, int nj, DATA_TYPE POLYBENCH_2D(B, NI, NJ, ni, nj), DATA_TYPE POLYBENCH_2D(B_outputFromGpu, NI, NJ, ni, nj))
{
    int i, j, fail, total;
    fail = 0;
    total = 0;

    // Compare outputs from CPU and GPU
    for (i = 1; i < (ni - 1); i++)
    {
        for (j = 1; j < (nj - 1); j++)
        {
            total++;
            if (percentDiff(B[i][j], B_outputFromGpu[i][j]) > PERCENT_DIFF_ERROR_THRESHOLD)
            {
                fail++;
            }
        }
    }

    // Print results
    printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f Percent: %d\n", PERCENT_DIFF_ERROR_THRESHOLD, fail);

    printf("Total number of comparations: %d\n", total);
    printf("Loop perforation rate: %f\n", LOOP_PERFORATION_RATE);
    printf("Block perforation rate: %f\n", BLOCK_PERFORATION_RATE);
}

void GPU_argv_init()
{
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, GPU_DEVICE);
    printf("setting device %d with name %s\n", GPU_DEVICE, deviceProp.name);
    hipSetDevice(GPU_DEVICE);
}

__global__ void convolution2D_kernel(int ni, int nj, DATA_TYPE *A, DATA_TYPE *B)
{
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;

    DATA_TYPE c11, c12, c13, c21, c22, c23, c31, c32, c33;

    c11 = +0.2;
    c21 = +0.5;
    c31 = -0.8;
    c12 = -0.3;
    c22 = +0.6;
    c32 = -0.9;
    c13 = +0.4;
    c23 = +0.7;
    c33 = +0.10;

    if ((i < _PB_NI - 1) && (j < _PB_NJ - 1) && (i > 0) && (j > 0))
    {
        B[i * NJ + j] = c11 * A[(i - 1) * NJ + (j - 1)] + c21 * A[(i - 1) * NJ + (j + 0)] + c31 * A[(i - 1) * NJ + (j + 1)] + c12 * A[(i + 0) * NJ + (j - 1)] + c22 * A[(i + 0) * NJ + (j + 0)] + c32 * A[(i + 0) * NJ + (j + 1)] + c13 * A[(i + 1) * NJ + (j - 1)] + c23 * A[(i + 1) * NJ + (j + 0)] + c33 * A[(i + 1) * NJ + (j + 1)];
    }
}

void convolution2DCuda(int ni, int nj, DATA_TYPE POLYBENCH_2D(A, NI, NJ, ni, nj), DATA_TYPE POLYBENCH_2D(B, NI, NJ, ni, nj),
                       DATA_TYPE POLYBENCH_2D(B_outputFromGpu, NI, NJ, ni, nj))
{
    DATA_TYPE *A_gpu;
    DATA_TYPE *B_gpu;

    hipMalloc((void **)&A_gpu, sizeof(DATA_TYPE) * NI * NJ);
    hipMalloc((void **)&B_gpu, sizeof(DATA_TYPE) * NI * NJ);
    hipMemcpy(A_gpu, A, sizeof(DATA_TYPE) * NI * NJ, hipMemcpyHostToDevice);

    dim3 block(ceil(DIM_THREAD_BLOCK_X * BLOCK_PERFORATION_RATE), ceil(DIM_THREAD_BLOCK_Y * BLOCK_PERFORATION_RATE));
    dim3 grid((size_t)ceil(((float)NI) / ((float)block.x) * GRID_PERFORATION_RATE), (size_t)ceil(((float)NJ) / ((float)block.y) * GRID_PERFORATION_RATE));

    /* Start timer. */
    GpuTimer gpuTimer;
    gpuTimer.Start();

    convolution2D_kernel<<<grid, block>>>(ni, nj, A_gpu, B_gpu);
    hipDeviceSynchronize();

    /* Stop and print timer. */
    gpuTimer.Stop();
    float elapsed_time = gpuTimer.Elapsed() / 1000;
    printf("GPU Time in seconds:\n");
    printf("%f\n", elapsed_time);

    hipMemcpy(B_outputFromGpu, B_gpu, sizeof(DATA_TYPE) * NI * NJ, hipMemcpyDeviceToHost);

    hipFree(A_gpu);
    hipFree(B_gpu);
}

/* DCE code. Must scan the entire live-out data.
   Can be used also to check the correctness of the output. */
static void print_array(int ni, int nj,
                        DATA_TYPE POLYBENCH_2D(B, NI, NJ, ni, nj))
{
    int i, j;

    for (i = 0; i < ni; i++)
        for (j = 0; j < nj; j++)
        {
            fprintf(stderr, DATA_PRINTF_MODIFIER, B[i][j]);
            if ((i * ni + j) % 20 == 0)
                fprintf(stderr, "\n");
        }
    fprintf(stderr, "\n");
}

int main(int argc, char *argv[])
{
    /* Retrieve problem size */
    int ni = NI;
    int nj = NJ;

    POLYBENCH_2D_ARRAY_DECL(A, DATA_TYPE, NI, NJ, ni, nj);
    POLYBENCH_2D_ARRAY_DECL(B, DATA_TYPE, NI, NJ, ni, nj);
    POLYBENCH_2D_ARRAY_DECL(B_outputFromGpu, DATA_TYPE, NI, NJ, ni, nj);

    // initialize the arrays
    init(ni, nj, POLYBENCH_ARRAY(A));

    GPU_argv_init();

    convolution2DCuda(ni, nj, POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(B), POLYBENCH_ARRAY(B_outputFromGpu));

#ifdef RUN_ON_CPU

    /* Start timer. */
    polybench_start_instruments;

    conv2D(ni, nj, POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(B));

    /* Stop and print timer. */
    printf("CPU Time in seconds:\n");
    polybench_stop_instruments;
    polybench_print_instruments;

    compareResults(ni, nj, POLYBENCH_ARRAY(B), POLYBENCH_ARRAY(B_outputFromGpu));

#else // prevent dead code elimination

    polybench_prevent_dce(print_array(ni, nj, POLYBENCH_ARRAY(B_outputFromGpu)));

#endif // RUN_ON_CPU

    POLYBENCH_FREE_ARRAY(A);
    POLYBENCH_FREE_ARRAY(B);
    POLYBENCH_FREE_ARRAY(B_outputFromGpu);

    return 0;
}

#include "../../utilities/polybench.c"